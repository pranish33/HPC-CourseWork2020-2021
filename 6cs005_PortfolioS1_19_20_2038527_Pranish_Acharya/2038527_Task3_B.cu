#include "hip/hip_runtime.h"
#include "lodepng.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime_api.h>
#include <time.h>

// nvcc -o CudaImageBlur 2038527_Task3_B.cu lodepng.cpp


__device__ unsigned int deviceWidth;

__device__ unsigned char getRed(unsigned char *image, unsigned int row, unsigned int col)
{
  unsigned int i = (row * deviceWidth * 4) + (col * 4);
  return image[i];
}

__device__ unsigned char getGreen(unsigned char *image, unsigned int row, unsigned int col)
{
  unsigned int i = (row * deviceWidth * 4) + (col * 4) + 1;
  return image[i];
}

__device__ unsigned char getBlue(unsigned char *image, unsigned int row, unsigned int col)
{
  unsigned int i = (row * deviceWidth * 4) + (col * 4) + 2;
  return image[i];
}

__device__ unsigned char getAlpha(unsigned char *image, unsigned int row, unsigned int col)
{
  unsigned int i = (row * deviceWidth * 4) + (col * 4) + 3;
  return image[i];
}

__device__ void setRed(unsigned char *image, unsigned int row, unsigned int col, unsigned char red)
{
  unsigned int i = (row * deviceWidth * 4) + (col * 4);
  image[i] = red;
}

__device__ void setGreen(unsigned char *image, unsigned int row, unsigned int col, unsigned char green)
{
  unsigned int i = (row * deviceWidth * 4) + (col * 4) + 1;
  image[i] = green;
}

__device__ void setBlue(unsigned char *image, unsigned int row, unsigned int col, unsigned char blue)
{
  unsigned int i = (row * deviceWidth * 4) + (col * 4) + 2;
  image[i] = blue;
}

__device__ void setAlpha(unsigned char *image, unsigned int row, unsigned int col, unsigned char alpha)
{
  unsigned int i = (row * deviceWidth * 4) + (col * 4) + 3;
  image[i] = alpha;
}

__global__ void changeImage(unsigned char* image, unsigned char* newImage, unsigned int *width){
  int row = blockIdx.x+1;
  int col = threadIdx.x+1;

  deviceWidth = *width;
  
  unsigned redTL, redTC, redTR;
  unsigned redL, redC, redR;
  unsigned redBL, redBC, redBR;
  unsigned newRed;

  unsigned greenTL, greenTC, greenTR;
  unsigned greenL, greenC, greenR;
  unsigned greenBL, greenBC, greenBR;
  unsigned newGreen;

  unsigned blueTL, blueTC, blueTR;
  unsigned blueL, blueC, blueR;
  unsigned blueBL, blueBC, blueBR;
  unsigned newBlue;
  
  setGreen(newImage, row, col, getGreen(image, row, col));
  setBlue(newImage, row, col, getBlue(image, row, col));
  setAlpha(newImage, row, col, 255);

  redTL = getRed(image, row - 1, col - 1);
  redTC = getRed(image, row - 1, col);
  redTR = getRed(image, row - 1, col + 1);

  redL = getRed(image, row, col - 1);
  redC = getRed(image, row, col);
  redR = getRed(image, row, col + 1);

  redBL = getRed(image, row + 1, col - 1);
  redBC = getRed(image, row + 1, col);
  redBR = getRed(image, row + 1, col + 1);

  newRed = (redTL+redTC+redTR+redL+redC+redR+redBL+redBC+redBR)/9;

  setRed(newImage, row, col, newRed);

  greenTL = getGreen(image, row - 1, col - 1);
  greenTC = getGreen(image, row - 1, col);
  greenTR = getGreen(image, row - 1, col + 1);

  greenL = getGreen(image, row, col - 1);
  greenC = getGreen(image, row, col);
  greenR = getGreen(image, row, col + 1);

  greenBL = getGreen(image, row + 1, col - 1);
  greenBC = getGreen(image, row + 1, col);
  greenBR = getGreen(image, row + 1, col + 1);

  newGreen = (greenTL+greenTC+greenTR+greenL+greenC+greenR+greenBL+greenBC+greenBR)/9;

  setGreen(newImage, row, col, newGreen);

  blueTL = getBlue(image, row - 1, col - 1);
  blueTC = getBlue(image, row - 1, col);
  blueTR = getBlue(image, row - 1, col + 1);

  blueL = getBlue(image, row, col - 1);
  blueC = getBlue(image, row, col);
  blueR = getBlue(image, row, col + 1);

  blueBL = getBlue(image, row + 1, col - 1);
  blueBC = getBlue(image, row + 1, col);
  blueBR = getBlue(image, row + 1, col + 1);

  newBlue = (blueTL+blueTC+blueTR+blueL+blueC+blueR+blueBL+blueBC+blueBR)/9;

  setBlue(newImage, row, col, newBlue);
}


int time_difference (struct timespec *start, struct timespec *finish, long long int *difference) {
	long long int ds = finish->tv_sec - start->tv_sec;
	long long int dn = finish->tv_nsec - start->tv_nsec;
	if (dn < 0) {
		ds--;
		dn += 1000000000;
	}
	*difference = ds * 1000000000 + dn;
	return! (*difference > 0);
}


int main(int argc, char **argv)
{
  struct timespec start, finish;
  long long int time_elapsed;
  clock_gettime(CLOCK_MONOTONIC, &start);
  
  unsigned char *image;
  const char *filename = argv[1];
  const char *newFileName = "filtered.png";
  unsigned char *newImage;
  unsigned int height = 0, width = 0;

  lodepng_decode32_file(&image, &width, &height, filename);
  newImage = (unsigned char *)malloc(height * width * 4 * sizeof(unsigned char));

  unsigned char * gpuImage;
  hipMalloc( (void**) &gpuImage, sizeof(char) * height*width*4); 
  hipMemcpy(gpuImage, image, sizeof(char) *  height*width*4, hipMemcpyHostToDevice);

  unsigned char * gpuNewImage;
  hipMalloc( (void**) &gpuNewImage, sizeof(char) * height*width*4);

  unsigned int* gpuWidth; 
  hipMalloc( (void**) &gpuWidth, sizeof(int));
  hipMemcpy(gpuWidth, &width, sizeof(int), hipMemcpyHostToDevice);

  printf("Image width = %d height = %d\n", width, height);

  changeImage<<<height-1,width-1>>>(gpuImage, gpuNewImage, gpuWidth);
  hipDeviceSynchronize();

  hipMemcpy(newImage, gpuNewImage, sizeof(char) * height * width * 4, hipMemcpyDeviceToHost);

  lodepng_encode32_file(newFileName, newImage, width, height);

  clock_gettime(CLOCK_MONOTONIC, &finish);
  time_difference(&start, &finish, &time_elapsed);
  printf("Time elapsed was %lldns or %0.9lfs\n", time_elapsed, (time_elapsed/1.0e9));
  return 0;
}


