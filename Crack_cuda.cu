#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include <crypt.h>
#include <time.h>
#include <math.h>
#include <malloc.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>



__device__ int is_a_match(char *attempt){
	char normal_password1[]="HP21";

	char *a=attempt;

	char *np1= normal_password1;

	
	while (*a==*np1){
		if(*a=='\0'){
			printf("password: %s\n", normal_password1);
			return 1;
		}
		a++;
		np1++;	
	}

	
	return 0;
}

__global__ void kernel(){
	int p=0, q=0 ;
	char password[4];
	password[3]='\0';

	int i= blockIdx.x+65;
	int j= threadIdx.x+65;

	char firstmatch=i;
	char secondmatch=j;
	password[0]= firstmatch;
	password[1]= secondmatch;
	for(p='0'; p<='9';p++){
		for(q='0'; q<='9';q++){
					password[2]=p;
					password[3]=q;

					if(is_a_match(password)){
					}					
					
				}
	}
}

int time_difference(struct timespec *start, 
                    struct timespec *finish, 
                    long long int *difference) {
  long long int ds =  finish->tv_sec - start->tv_sec; 
  long long int dn =  finish->tv_nsec - start->tv_nsec; 

  if(dn < 0 ) {
    ds--;
    dn += 1000000000; 
  } 
  *difference = ds * 1000000000 + dn;
  return !(*difference > 0);
}

int main(int argc, char *argv[]){
  struct timespec start, finish;   
  long long int time_elapsed;
  clock_gettime(CLOCK_MONOTONIC, &start);

  kernel <<<26,26>>>();
  hipDeviceSynchronize();

  clock_gettime(CLOCK_MONOTONIC, &finish);
  time_difference(&start, &finish, &time_elapsed);
  printf("Time elapsed was %lldns or %0.9lfs\n", time_elapsed,
                                         (time_elapsed/1.0e9));

  return 0;
}
