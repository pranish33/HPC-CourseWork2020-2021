
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

__device__ char* CudaCrypt(char* rawPassword){

	 
	 char * newPassword = (char *) malloc(sizeof(char) * 11);

	newPassword[0] = rawPassword[0] + 2;
	newPassword[1] = rawPassword[0] - 2;
	newPassword[2] = rawPassword[0] + 1;
	newPassword[3] = rawPassword[1] + 3;
	newPassword[4] = rawPassword[1] - 3;
	newPassword[5] = rawPassword[1] - 1;
	newPassword[6] = rawPassword[2] + 2;
	newPassword[7] = rawPassword[2] - 2;
	newPassword[8] = rawPassword[3] + 4;
	newPassword[9] = rawPassword[3] - 4;
	newPassword[10] = '\0';

	for(int i =0; i<10; i++){
		if(i >= 0 && i < 6){ 
			if(newPassword[i] > 122){ 
				newPassword[i] = (newPassword[i] - 122) + 97;
			}else if(newPassword[i] < 97){
				newPassword[i] = (97 - newPassword[i]) + 97;
			}
		}else{ 
			if(newPassword[i] > 57){
				newPassword[i] = (newPassword[i] - 57) + 48;
			}else if(newPassword[i] < 48){
				newPassword[i] = (48 - newPassword[i]) + 48;
			}
		}
		
	}
         
       
        
	return newPassword; 
}

__device__ int is_password(char* Encrypted){
      
       char Password[]="pa15";
       
       char *a=Encrypted;
       
       char *p=CudaCrypt(Password);
  
	while (*a == *p){
	
	if (*a == '\0')
	{
	printf("Encrypted Password: %s\n",Encrypted);
	printf("Password Found: %s\n",Password);
	break;
	}
	a++;
	p++;
	}
	return 0;
}

__global__ void crack(char * alphabet, char * numbers){ 
    
   char genRawPass[4];

   genRawPass[0] = alphabet[blockIdx.x];
   genRawPass[1] = alphabet[blockIdx.y];

   genRawPass[2] = numbers[threadIdx.x];
   genRawPass[3] = numbers[threadIdx.y];
   
   char *generated=CudaCrypt(genRawPass);
	
   is_password(generated);
   
  
        

}

int time_difference(struct timespec *start, 
                    struct timespec *finish, 
                    long long int *difference) {
  long long int ds =  finish->tv_sec - start->tv_sec; 
  long long int dn =  finish->tv_nsec - start->tv_nsec; 

  if(dn < 0 ) {
    ds--;
    dn += 1000000000; 
  } 
  *difference = ds * 1000000000 + dn;
  return !(*difference > 0);
}

int main(int argc, char ** argv){
  struct timespec start, finish;   
  long long int time_elapsed;
  clock_gettime(CLOCK_MONOTONIC, &start);

  char cpuAlphabet[26] =  {'a','b','c','d','e','f','g','h','i','j','k','l','m','n','o','p','q','r','s','t','u','v','w','x','y','z'};
  char cpuNumbers[26] = {'0','1','2','3','4','5','6','7','8','9'};

  char * gpuAlphabet;
  hipMalloc( (void**) &gpuAlphabet, sizeof(char) * 26); 
  hipMemcpy(gpuAlphabet, cpuAlphabet, sizeof(char) * 26, hipMemcpyHostToDevice);

  char * gpuNumbers;
  hipMalloc( (void**) &gpuNumbers, sizeof(char) * 26); 
  hipMemcpy(gpuNumbers, cpuNumbers, sizeof(char) * 26, hipMemcpyHostToDevice);

  crack<<< dim3(26,26,1), dim3(10,10,1) >>>( gpuAlphabet, gpuNumbers);  
  
  hipDeviceSynchronize();
  
  clock_gettime(CLOCK_MONOTONIC, &finish);
  time_difference(&start, &finish, &time_elapsed);
  printf("Time elapsed was %lldns or %0.9lfs\n", time_elapsed, (time_elapsed/1.0e9));
  return 0;
}













